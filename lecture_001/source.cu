#include "hip/hip_runtime.h"
#include <torch/torch.h>
#include <torch/types.h>

__global__ void square_matrix_kernel(const float* matrix, float* result, int width, int height) {

}

torch::Tensor square_matrix(torch::Tensor matrix) {
    const auto height = matrix.size(0);
    const auto width = matrix.size(1);
    auto result = torch::empty_like(matrix);
    dim3 threads_per_block(16,16);
    dim3 number_of_blocks(
        (width + threads_per_block.x - 1) / threads_per_block.x, 
        (height + threads_per_block.y - 1) / threads_per_block.y
    );

}
